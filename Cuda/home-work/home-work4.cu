#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include "../utils/cuda_utils.cuh"

using namespace std;

__global__ void monteCarlo(int* inCircle, int maxSize)
{
	hiprandState state = initState();

	float x = getRandFloat(&state) * 1;
	float y = getRandFloat(&state) * 1;

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1) {
		atomicAdd((inCircle + (getCurrentThreadNumber())), 1);
	}
}

int homeWork4() {
	dim3 gridSize(256);
	dim3 blockSize(256);

	int threadsCount = blockSize.x * blockSize.y * blockSize.z;

	int * hostInCircle = new int[threadsCount];

	int hostInSquare = gridSize.x * gridSize.y * gridSize.z * blockSize.x * blockSize.y * blockSize.z;

	int * deviceInCircle;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, threadsCount * sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <gridSize, blockSize >> > (deviceInCircle, threadsCount);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(hostInCircle, deviceInCircle, threadsCount * sizeof(int), hipMemcpyDeviceToHost);

	int hostInCircleTotal = 0;

	for (int i = 0; i < threadsCount; i++) {
		hostInCircleTotal += *(hostInCircle + i);
	}

	float pi = (4 * float(hostInCircleTotal) / float(hostInSquare));

	printf("Pi: %f\n", pi);

	float error = abs(4 * atan(1) - pi);
	printf("\nError: %f\n", error);

	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipFree(deviceInCircle);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}