#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include "../utils/cuda_utils.cuh"

using namespace std;

__global__ void monteCarlo(int* inCircle, int* inSquare, int maxSize)
{
	hiprandState state = initState();

	float x = getRandFloat(&state) * 1;
	float y = getRandFloat(&state) * 1;

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1) {
		atomicAdd((inCircle + (getCurrentThreadNumber())), 1);
	}
	else {
		atomicAdd((inSquare + (getCurrentThreadNumber())), 1);
	}
}

int homeWork4() {
	dim3 gridSize(400);
	dim3 blockSize(1024);

	int threadsCount = blockSize.x * blockSize.y * blockSize.z;

	int * hostInCircle = new int[threadsCount];
	int * hostInSquare = new int[threadsCount];

	int * deviceInCircle;
	int * deviceInSquare;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, threadsCount * sizeof(int));
	hipMalloc((void**)& deviceInSquare, threadsCount * sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <gridSize, blockSize >> > (deviceInCircle, deviceInSquare, threadsCount);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(hostInCircle, deviceInCircle, threadsCount * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hostInSquare, deviceInSquare, threadsCount * sizeof(int), hipMemcpyDeviceToHost);


	int hostInCircleTotal = 0;
	int hostInSquareTotal = 0;

	for (int i = 0; i < threadsCount; i++) {
		hostInCircleTotal += *(hostInCircle + i);
		hostInSquareTotal += *(hostInSquare + i);
	}

	float pi = (4 * float(hostInCircleTotal) / (float(hostInSquareTotal) + float(hostInCircleTotal)));

	printf("Pi: %f\n", pi);

	float error = abs(4 * atan(1) - pi);
	printf("\nError: %f\n", error);

	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipFree(deviceInCircle);
	hipFree(deviceInSquare);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}