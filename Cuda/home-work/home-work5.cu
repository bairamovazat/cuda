#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include "../utils/cuda_utils.cuh"
#include <cmath>

using namespace std;

#define BASE_TYPE float
#define M_PI 3.141592653

__global__ void calculateVector(int row, float* sourceAMatrix, float* targetBMatrix, float* buffetProjMatrix, int rows, int columns) {
	int vectorNumber = threadIdx.x;
	int vectorElement = threadIdx.y;
	float* bVector = targetBMatrix + (row * columns);
	float* aVector = sourceAMatrix + (row * columns);
	if (vectorNumber == 0) {
		atomicAdd(bVector + vectorElement, *(aVector + vectorElement));
	}
	else if (vectorNumber > row) {
		//Ничего не делаем, дальше элементы не считаем
	}
	else {
		float upper = *(buffetProjMatrix + vectorNumber);
		float lower = *(buffetProjMatrix + columns + vectorNumber);
		printf("Upper: %f, Lower: %f, %d, %d, %d\n", upper, lower, row, vectorNumber, vectorElement);
		atomicAdd(bVector + vectorElement, (*(aVector + vectorElement) * -upper / lower));
	}

}

__global__ void calculateProj(int row, float* sourceAMatrix, float* targetBMatrix, float* buffetProjMatrix, int rows, int columns) {
	int bVectorIndex = threadIdx.x - 1;
	int vectorElement = threadIdx.y;
	//Если 0, то верхнее <a,b>
	//Если 1, то нижнее  <b,b>
	int lowerExpr = blockIdx.x;

	float* target = buffetProjMatrix + (columns * lowerExpr) + (bVectorIndex + 1);

	float* result = 0;

	//Если это нулевой элемент proj(b-1,a_row)
	if (bVectorIndex == -1)
	{
		//atomicAdd(target, *(sourceAMatrix + (row * columns) + vectorElement));
	}
	else if (bVectorIndex >= row) {
		//Ничего не делаем, дальше элементы не считаем
	}
	else {
		float* vectorA;

		if (lowerExpr == 1) {
			vectorA = targetBMatrix + (bVectorIndex * columns);
		}
		else {
			vectorA = sourceAMatrix + (row * columns);
		}

		float* vectorB = targetBMatrix + (bVectorIndex * columns);

		atomicAdd(target, (*(vectorB + vectorElement) * *(vectorA + vectorElement)));
	}

	return;
}

void printMatrix(float* matrix, int rows, int columns) {
	for (int row = 0; row < rows; row++) {
		for (int column = 0; column < columns; column++) {
			cout << *(matrix + (row * columns) + column) << ", ";
		}
		cout << endl;
	}
}

void fillMatrix(float* matrix, int rows, int columns, bool allZero, bool upperTreangle) {
	for (int row = 0; row < rows; row++) {
		for (int column = 0; column < columns; column++) {
			if (upperTreangle && row <= column) {
				*(matrix + (row * columns) + column) = 1.0;
			}
			else if(upperTreangle && row < column || allZero) {
				*(matrix + (row * columns) + column) = 0.0;
			}
		}
	}
}

int homeWork5() {
	size_t rows = 3;
	size_t columns = 3;

	dim3 gridSizeProj(2);
	dim3 gridSize(1);

	dim3 blockSize(rows, columns);

	int elementCount = rows * columns;

	float* sourceAMatrix = new float[elementCount];

	fillMatrix(sourceAMatrix, rows, columns, true, false);
	printMatrix(sourceAMatrix, rows, columns);
	cout << "--------------------------" << endl;
	fillMatrix(sourceAMatrix, rows, columns, false, true);
	printMatrix(sourceAMatrix, rows, columns);
	cout << "--------------------------" << endl;

	float* deviceSourceAMatrix;
	hipMalloc((void**)&deviceSourceAMatrix, elementCount * sizeof(float));

	float* targetBMatrix = new float[elementCount];
	fillMatrix(targetBMatrix, rows, columns, true, false);
	float* deviceTargetBMatrix;
	hipMalloc((void**)&deviceTargetBMatrix, elementCount * sizeof(float));

	float* buffetProjMatrix = new float[columns * 2];
	fillMatrix(targetBMatrix, 2, columns, true, false);
	float* deviceBuffetProjMatrix;
	hipMalloc((void**)&deviceBuffetProjMatrix, columns * 2 * sizeof(float));

	float* buffetProjMatrixToPrint = new float[columns * 2];

	hipMemcpy(deviceSourceAMatrix, sourceAMatrix, elementCount * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 0; i < rows; i++) {
		hipMemcpy(deviceBuffetProjMatrix, buffetProjMatrix, columns * 2 * sizeof(float), hipMemcpyHostToDevice);

		calculateProj << <gridSizeProj, blockSize >> > (i, deviceSourceAMatrix, deviceTargetBMatrix, deviceBuffetProjMatrix, rows, columns);
		hipDeviceSynchronize();
		calculateVector << <gridSize, blockSize >> > (i, deviceSourceAMatrix, deviceTargetBMatrix, deviceBuffetProjMatrix, rows, columns);

		hipMemcpy(buffetProjMatrixToPrint, deviceBuffetProjMatrix, 2 * columns * sizeof(float), hipMemcpyDeviceToHost);
		printMatrix(buffetProjMatrixToPrint, 2, columns);
		cout << "--------------------------" << endl;
		hipMemcpy(targetBMatrix, deviceTargetBMatrix, elementCount * sizeof(float), hipMemcpyDeviceToHost);
		printMatrix(targetBMatrix, rows, columns);
		cout << "--------------------------" << endl;

	}

	hipMemcpy(targetBMatrix, deviceTargetBMatrix, elementCount * sizeof(float), hipMemcpyDeviceToHost);

	printMatrix(targetBMatrix, rows, columns);

	hipFree(deviceSourceAMatrix);
	hipFree(deviceTargetBMatrix);
	hipFree(deviceBuffetProjMatrix);

	delete[] sourceAMatrix;
	delete[] targetBMatrix;
	delete[] buffetProjMatrix;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	return 0;
}