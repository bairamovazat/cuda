#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include <cmath>
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "../utils/cuda_utils.cuh"

using namespace std;


__global__ void deviceVectorRateWithShared(float** firstMatrix, float* target) {
	int threadId = threadIdx.x;
	int vectorId = blockIdx.x * blockDim.x + threadId;
	extern __shared__ float sharedMemory[];

	//Это эквивалентно записи просто в локальную переменную
	sharedMemory[threadId] = firstMatrix[vectorId][0];

	__syncthreads();

	atomicAdd(target, sharedMemory[threadId] * sharedMemory[threadId]);
}

__global__ void deviceVectorRate(float** firstMatrix, float* target) {
	int threadId = threadIdx.x;
	int vectorId = blockIdx.x * blockDim.x + threadId;
	atomicAdd(target, firstMatrix[vectorId][0] *  firstMatrix[vectorId][0]);
}

void vectorRate(float** hostFirstMatrix, float* result, unsigned rows) {
	float** deviceFirstArray = initArrayToDevice(rows, 1);
	сopyArrayToDevice(hostFirstMatrix, deviceFirstArray, rows, 1);

	float* deviceResult;
	hipMalloc((void**) &deviceResult, sizeof(float));
	hipMemcpy(deviceResult, result, sizeof(float), hipMemcpyHostToDevice);

	dim3 gridSize(rows / 16);
	dim3 blockSize(16);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//В первый блок будем записывать результат
	deviceVectorRateWithShared << <gridSize, blockSize, (16) * sizeof(float) >> > (deviceFirstArray, deviceResult);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("Вычисление элемента через shared: %f millseconds\n", KernelTime);

	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
	//В первый блок будем записывать результат
	deviceVectorRate << <gridSize, blockSize, (16) * sizeof(float) >> > (deviceFirstArray, deviceResult);
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	float KernelTime2;
	hipEventElapsedTime(&KernelTime, start2, stop2);
	printf("Вычисление элемента: %f millseconds\n", KernelTime2);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

	hipMemcpy(result, deviceResult, sizeof(float), hipMemcpyDeviceToHost);
}

int homeWork7() {
	unsigned arrayRows = 32;
	unsigned arrayColumns = 1;

	unsigned secondArrayRows = 32;
	unsigned secondArrayColumns = 1;

	float** hostFirstArray = initArrayWithGenerator(arrayRows, arrayColumns, [](int i, int j, int rows, int columns) {return i * columns + j; });
	float result = 0;
	float* target = &result;

	//cout << "---------Норма--------\n";
	//printMatrix(hostFirstArray, arrayRows, arrayColumns);
	//cout << "----------------------\n";

	vectorRate(hostFirstArray, target, arrayRows);
	//cout << "----------------------\n";
	cout << *target;

	return 0;
}