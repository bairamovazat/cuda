#include "hip/hip_runtime.h"
//����������� ���������
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include <cmath>
#include <hip/device_functions.h>
#include <assert.h>
#include <hip/hip_texture_types.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "../utils/cuda_utils.cuh"

using namespace std;

//#define VECTOR_SIZE 4096
#define VECTOR_SIZE 256
#define GRID_SIZE 32

__device__ __constant__ float constFirstVector[VECTOR_SIZE];
__device__ __constant__ float constSecondVector[VECTOR_SIZE];

texture<float> firstTexture;
texture<float> secondTexture;

__global__ void deviceMultiply(float* firstVector, float* secondVector, float* result) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	atomicAdd(result, *(firstVector + id) * *(secondVector + id));
}

void multiply(float* firstVector, float* secondVector, float* result) {

	float* deviceFirstVector;
	hipMalloc(&deviceFirstVector, VECTOR_SIZE * sizeof(float));
	hipMemcpy(deviceFirstVector, firstVector, VECTOR_SIZE * sizeof(float), hipMemcpyHostToDevice);

	float* deviceSecondVector;
	hipMalloc(&deviceSecondVector, VECTOR_SIZE * sizeof(float));
	hipMemcpy(deviceSecondVector, secondVector, VECTOR_SIZE * sizeof(float), hipMemcpyHostToDevice);

	float* deviceResult;
	hipMalloc(&deviceResult, sizeof(float));
	hipMemcpy(deviceResult, result, sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	deviceMultiply << <VECTOR_SIZE / GRID_SIZE, GRID_SIZE >> > (deviceFirstVector, deviceSecondVector, deviceResult);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(result, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("���������� ���������� ������������: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
}

__global__ void deviceMultiply(float* result) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	atomicAdd(result, constFirstVector[id] * constSecondVector[id]);
}

void constMultiply(float* firstVector, float* secondVector, float* result) {
	hipMemcpyToSymbol(HIP_SYMBOL(constFirstVector), firstVector, VECTOR_SIZE * sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(constSecondVector), secondVector, VECTOR_SIZE * sizeof(float));

	float* deviceResult;
	hipMalloc(&deviceResult, sizeof(float));
	hipMemcpy(deviceResult, result, sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	deviceMultiply << <VECTOR_SIZE / GRID_SIZE, GRID_SIZE >> > (deviceResult);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(result, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("���������� ���������� ������������ ����� __constant__: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
}

void printVector(float* vector, int size) {
	for (int i = 0; i < size; i++) {
		printf("%f, ", *(vector + i));
	}
	printf("\n");
}
void constMemoryTest() {
	float* firstVector = new float[VECTOR_SIZE];
	float* secondVector = new float[VECTOR_SIZE];

	for (int i = 0; i < VECTOR_SIZE; i++) {
		*(firstVector + i) = i;
		*(secondVector + i) = i;
	}
	printVector(firstVector, VECTOR_SIZE);
	printVector(secondVector, VECTOR_SIZE);
	cout << "-----------------constMultiply-----------------\n";
	float result = 0;
	constMultiply(firstVector, secondVector, &result);
	printf("result: %f\n", result);
	cout << "-----------------multiply-----------------\n";
	float simpleResult = 0;
	multiply(firstVector, secondVector, &simpleResult);
	printf("result: %f\n", simpleResult);
}
__global__ void textureDeviceMultiply(float* result) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float x2 = tex1D(secondTexture, float(id));
	float x1 = tex1D(firstTexture, float(id));

	atomicAdd(result, x1 * x1);
}

void textureMultiply(float* hostFirstVector, float* hostSecondVector, float* result, int size) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(size, 0, 0, 0, hipChannelFormatKindFloat);

	hipArray* firstDeviceCudaArray;
	hipMallocArray(&firstDeviceCudaArray, &firstTexture.channelDesc, size, 1);
	hipMemcpyToArray(firstDeviceCudaArray, 0, 0, hostFirstVector, sizeof(float) * size, hipMemcpyHostToDevice);
	hipBindTextureToArray(firstTexture, firstDeviceCudaArray, firstTexture.channelDesc);

	float* secondDeviceArray;
	hipMalloc((void**)&secondDeviceArray, size * sizeof(float));
	hipMemcpy(secondDeviceArray, hostSecondVector, size * sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, secondTexture, secondDeviceArray, firstTexture.channelDesc, size * sizeof(float));

	float* deviceResult;
	hipMalloc(&deviceResult, sizeof(float));
	hipMemcpy(deviceResult, result, sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	textureDeviceMultiply << <VECTOR_SIZE / GRID_SIZE, GRID_SIZE >> > (deviceResult);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(result, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("���������� ���������� ������������ ����� ���������� ������ ��������� hipArray � �������� ������: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
}

void textureMemoryTest() {
	float* firstVector = new float[VECTOR_SIZE];
	float* secondVector = new float[VECTOR_SIZE];

	for (int i = 0; i < VECTOR_SIZE; i++) {
		*(firstVector + i) = i;
		*(secondVector + i) = i;
	}
	printVector(firstVector, VECTOR_SIZE);
	printVector(secondVector, VECTOR_SIZE);

	float result = 0;
	textureMultiply(firstVector, secondVector, &result, VECTOR_SIZE);
	printf("result: %f\n", result);
}


int homeWork8() {
	cout << "-----------------constMemoryTest-----------------\n";
	constMemoryTest();
	cout << "-----------------textureMemoryTest-----------------\n";
	textureMemoryTest();
	return 0;
}