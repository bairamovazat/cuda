#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
using namespace std;

__device__ void randomInt2(float *i, float *j) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init(1234, idx, 0, &state);
	float ranv1 = hiprand_uniform(&state);
	float ranv2 = hiprand_uniform(&state);
	*i = ranv1 * 1;
	*j = ranv2 * 1;
}

__global__ void monteCarlo(int* inCircle, int* inSquare)
{
	float x, y;
	randomInt2(&x, &y);

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1.0) {
		atomicAdd(inCircle, 1);
	}
	else {
		atomicAdd(inSquare, 1);
	}
}

int homeWork4() {
	dim3 gridSize(300);
	dim3 blockSize(50);
	int hostInCircle = 0;
	int hostInSquare = 0;
	int *deviceInCircle;
	int *deviceInSquare;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, sizeof(int));
	hipMalloc((void**)& deviceInSquare, sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <300, 200 >> > (deviceInCircle, deviceInSquare);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(&hostInCircle, deviceInCircle, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&hostInSquare, deviceInSquare, sizeof(int), hipMemcpyDeviceToHost);

	printf("hostInCircle: %d\n", hostInCircle);
	printf("hostInSquare: %d\n", hostInSquare);

	float pi = (4 * float(hostInCircle) / (float(hostInSquare) + float(hostInCircle)));
	printf("%f\n", pi);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}