#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
using namespace std;

#define WARP_SIZE 32;

__device__ int getCurrentThreadId() {
	int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
	int threadId = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ hiprandState initState() {
	int id = getCurrentThreadId();
	hiprandState state;
	hiprand_init(1234, id, 0, &state);
	return state;
}

__device__ float getRandFloat(hiprandState *state) {
	return hiprand_uniform(state);
}

__device__ float getRandFloat() {
	hiprandState state = initState();
	return getRandFloat(&state);
}

__global__ void monteCarlo(int* inCircle, int* inSquare)
{
	hiprandState state = initState();

	float x = getRandFloat(&state) * 1;
	float y = getRandFloat(&state) * 1;

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1) {
		atomicAdd(inCircle, 1);
	}
	else {
		atomicAdd(inSquare, 1);
	}
}

int homeWork4() {
	dim3 gridSize(300);
	dim3 blockSize(50);
	int hostInCircle = 0;
	int hostInSquare = 0;
	int *deviceInCircle;
	int *deviceInSquare;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, sizeof(int));
	hipMalloc((void**)& deviceInSquare, sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <300, 200 >> > (deviceInCircle, deviceInSquare);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(&hostInCircle, deviceInCircle, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&hostInSquare, deviceInSquare, sizeof(int), hipMemcpyDeviceToHost);

	printf("hostInCircle: %d\n", hostInCircle);
	printf("hostInSquare: %d\n", hostInSquare);

	float pi = (4 * float(hostInCircle) / (float(hostInSquare) + float(hostInCircle)));
	printf("%f\n", pi);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}