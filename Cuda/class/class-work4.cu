#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include "../utils/cuda_utils.cuh"

using namespace std;

__global__ void monteCarlo(int* inCircle, int* inSquare)
{
	hiprandState state = initState();

	float x = getRandFloat(&state) * 1;
	float y = getRandFloat(&state) * 1;

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1) {
		atomicAdd(inCircle, 1);
	}
	else {
		atomicAdd(inSquare, 1);
	}
}

int classWork4() {
	dim3 gridSize(1024);
	dim3 blockSize(2048);

	int hostInCircle = 0;
	int hostInSquare = 0;

	int * deviceInCircle;
	int * deviceInSquare;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, sizeof(int));
	hipMalloc((void**)& deviceInSquare, sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <gridSize, blockSize >> > (deviceInCircle, deviceInSquare);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(&hostInCircle, deviceInCircle, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&hostInSquare, deviceInSquare, sizeof(int), hipMemcpyDeviceToHost);


	float pi = (4 * float(hostInCircle) / (float(hostInSquare) + float(hostInCircle)));

	printf("Pi: %f\n", pi);
	
	float error = abs(4 * atan(1) - pi);
	printf("\nError: %f\n", error);


	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipFree(deviceInCircle);
	hipFree(deviceInSquare);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}