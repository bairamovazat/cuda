#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include "../utils/cuda_utils.cuh"

using namespace std;

__global__ void monteCarlo(int* inCircle)
{
	hiprandState state = initState();

	float x = getRandFloat(&state) * 1;
	float y = getRandFloat(&state) * 1;

	if (sqrt(float(pow(x,2) + pow(y,2))) < 1) {
		atomicAdd(inCircle, 1);
	}
}

int classWork4() {
	dim3 gridSize(256);
	dim3 blockSize(256);

	int hostInCircle = 0;
	int hostInSquare = gridSize.x * gridSize.y * gridSize.z * blockSize.x * blockSize.y * blockSize.z;
	int * deviceInCircle;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& deviceInCircle, sizeof(int));

	hipEventRecord(start, 0);

	monteCarlo << <gridSize, blockSize >> > (deviceInCircle);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(&hostInCircle, deviceInCircle, sizeof(int), hipMemcpyDeviceToHost);


	float pi = (4 * float(hostInCircle) / float(hostInSquare));

	printf("Pi: %f\n", pi);
	
	float error = abs(4 * atan(1) - pi);
	printf("\nError: %f\n", error);


	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %f millseconds\n", KernelTime);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ", hipGetErrorString(err));

	hipFree(deviceInCircle);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}