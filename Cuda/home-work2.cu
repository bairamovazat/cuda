#include "hip/hip_runtime.h"
﻿//подключение библиотек
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
using namespace std;

#define N 1
#define M 200

__device__ void randomInt(float *i, float *j) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	hiprand_init(1234, idx, 0, &state);
	float ranv1 = hiprand_uniform(&state);
	float ranv2 = hiprand_uniform(&state);
	*i = ranv1 * N;
	*j = ranv2 * N;
}

__global__ void monteCarloZuf(int* res1, int *res2) {
	float i, j;
	randomInt(&i, &j);
	if (sqrt(float(i*i+j*j))<N)// ïðîâåðêà óñëîâèÿ 
	{
		atomicAdd(res1, 1);
	}
	else {
		atomicAdd(res2, 1);
	}
}

int homeWork2() {
	int host_a;
	int host_b;
	int* dev_a;
	int* dev_b;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)& dev_a, sizeof(int));
	hipMalloc((void**)& dev_b, sizeof(int));
	hipMemset(dev_a, 0, sizeof(int));

	hipEventRecord(start, 0);

	monteCarloZuf << <300,32 >> >(dev_a, dev_b);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTme: %f millseconds\n", KernelTime);

	hipMemcpy(&host_a, dev_a, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&host_b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
	float res = 4* host_a / float(host_a + host_b);
	printf("Value PI: %f\n", res);
	float error = abs(4 * atan(1) - res);
	printf("\nError: %f\n", error);

	//for (int i = 0; i < N; i++) { 
	//	if (host_a != N) 
	//		printf("error [%d] -> %d\n", i, host_a[i]); 
	//}

	// Ïðîâåðêà íà îøèáêó âûïîëíåíèÿ ïðîãðàììû íà äåâàéñå
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("%s ", hipGetErrorString(err));
	hipFree(dev_a);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
 }