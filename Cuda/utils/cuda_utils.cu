#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

using namespace std;

//��������� ���������� ������������� ����� �� ���� �����
__device__ int getCurrentThreadId() {
	const unsigned long long int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;

	const unsigned long long int threadId = blockId * blockDim.x + threadIdx.x;

	return threadId;
}

//�������� ������� ������� ����� ����� (2 * 2 * 2 ) - 0...7
//���� ���������� ��� 3 ������ ������
__device__ int getCurrentThreadNumber() {
	return threadIdx.x;
}

__device__ hiprandState initState() {
	int id = getCurrentThreadId();
	hiprandState state;
	hiprand_init(1234, id, 0, &state);
	return state;
}

__device__ float getRandFloat(hiprandState* state) {
	return hiprand_uniform(state);
}

__device__ float getRandFloat() {
	hiprandState state = initState();
	return getRandFloat(&state);
}

//template <typename T>
//T** initArrayToDevice(unsigned row, unsigned column) {
//	T** array;
//	hipMalloc((void**) & array, row * sizeof(T*));
//	for (int i = 0; i < row; i++) {   // (3)
//		T* arrayElement;
//		hipMalloc((void**)&arrayElement, column * sizeof(T));
//		array[i] = arrayElement;     // ������������� ����������
//	}
//
//	return array;
//}
//
//template <typename T>
//void �opyArrayToDevice(T** hostArray, T** deviceArray, unsigned row, unsigned column) {
//	for (int i = 0; i < row; i++) {   // (3)
//		hipMemcpy(deviceArray[i], hostArray[i], column * sizeof(T), hipMemcpyHostToDevice);
//	}
//}
//
//template <typename T>
//void �opyArrayToHost(T** hostArray, T** deviceArray, unsigned row, unsigned column) {
//	for (int i = 0; i < row; i++) {   // (3)
//		hipMemcpy(hostArray[i], deviceArray[i], column * sizeof(T), hipMemcpyDeviceToHost);
//	}
//}
//
//template <typename T>
//T** initAndCopyArrayToDevice(T** hostArray, unsigned row, unsigned column) {
//	T** deviceArray = initArrayToDevice<T>(row, column);
//	�opyArrayToDevice(hostArray, deviceArray, row, column);
//	return deviceArray;
//}
//
//template <typename T>
//void freeArray(T** deviceArray) {
//	hipFree(deviceArray);
//}
//
//
//int generatorZero(int i, int j, int row, int column) {
//	return 0;
//}
//
//template <typename T>
//T** initArrayWithGenerator(unsigned row, unsigned column, int (generateValue)(int, int, int, int)) {
//	// ��������
//	T** array = new T * [row];    // ������ ���������� (2)
//	for (int i = 0; i < row; i++) {   // (3)
//		array[i] = new T[column];     // ������������� ����������
//		for (int j = 0; j < column; j++) {
//			array[i][j] = (generateValue)(i, j, row, column);
//		}
//	}
//	return array;
//}
//
//template <typename T>
//T** initArray(unsigned row, unsigned column) {
//	return initArrayWithGenerator<T>(row, column, generatorZero);
//}
//
//
//template <typename T>
//void freeArray(T** array, unsigned row, unsigned column) {
//	for (int i = 0; i < row; i++) {
//		delete[] array[i];
//	}
//	delete[] array;
//}


float** initArrayToDevice(unsigned row, unsigned column) {
	float** tempArray = new float* [row];
	float** array;
	hipMalloc((void**) & array, row * sizeof(float*));
	for (int i = 0; i < row; i++) {
		hipMalloc(&tempArray[i], column * sizeof(float));
	}
	hipMemcpy(array, tempArray, row * sizeof(float*), hipMemcpyHostToDevice);

	return array;
}

void �opyArrayToDevice(float** hostArray, float** deviceArray, unsigned row, unsigned column) {
	float** tempArray = new float* [row];
	hipMemcpy(tempArray, deviceArray, row * sizeof(float*), hipMemcpyDeviceToHost);
	for (int i = 0; i < row; i++) {
		hipMemcpy(tempArray[i], hostArray[i], column * sizeof(float), hipMemcpyHostToDevice);
	}
}

void �opyArrayToHost(float** hostArray, float** deviceArray, unsigned row, unsigned column) {
	float** tempArray = new float* [row];
	hipMemcpy(tempArray, deviceArray, row * sizeof(float*), hipMemcpyDeviceToHost);

	for (int i = 0; i < row; i++) {
		hipMemcpy(hostArray[i], tempArray[i], column * sizeof(float), hipMemcpyDeviceToHost);
	}
}

float** initAndCopyArrayToDevice(float** hostArray, unsigned row, unsigned column) {
	float** deviceArray = initArrayToDevice(row, column);
	�opyArrayToDevice(hostArray, deviceArray, row, column);
	return deviceArray;
}

void freeArray(float** deviceArray) {
	hipFree(deviceArray);
}

int generatorZero(int i, int j, int row, int column) {
	return 0;
}

float** initArrayWithGenerator(unsigned row, unsigned column, int (generateValue)(int, int, int, int)) {
	// ��������
	float** array = new float* [row];    // ������ ���������� (2)
	for (int i = 0; i < row; i++) {   // (3)
		array[i] = new float[column];     // ������������� ����������
		for (int j = 0; j < column; j++) {
			array[i][j] = (generateValue)(i, j, row, column);
		}
	}
	return array;
}

float** initArray(unsigned row, unsigned column) {
	return initArrayWithGenerator(row, column, generatorZero);
}

void freeArray(float** array, unsigned row, unsigned column) {
	for (int i = 0; i < row; i++) {
		delete[] array[i];
	}
	delete[] array;
}

void printMatrix(float** matrix, unsigned rows, unsigned columns) {
	for (int row = 0; row < rows; row++) {
		float* element = *(matrix + row);
		for (int column = 0; column < columns; column++) {
			cout << *(element + column) << ", ";
		}
		cout << endl;
	}
}