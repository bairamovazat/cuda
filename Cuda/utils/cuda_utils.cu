#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

//��������� ���������� ������������� ����� �� ���� �����
__device__ unsigned long long getCurrentThreadId() {
	const unsigned long long int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z; 

	const unsigned long long int threadId = blockId * blockDim.x + threadIdx.x;

	return threadId;
}

//�������� ������� ������� ����� ����� (2 * 2 * 2 ) - 0...7
//���� ���������� ��� 3 ������ ������
__device__ int getCurrentThreadNumber() {
	return threadIdx.x;
}

__device__ hiprandState initState() {
	int id = getCurrentThreadId();
	hiprandState state;
	hiprand_init(1234, id, 0, &state);
	return state;
}

__device__ float getRandFloat(hiprandState* state) {
	return hiprand_uniform(state);
}

__device__ float getRandFloat() {
	hiprandState state = initState();
	return getRandFloat(&state);
}


