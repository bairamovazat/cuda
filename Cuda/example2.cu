#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void warpTest()
{
	printf("BlockId: %d, ThreadId: %d\n", blockIdx.x, threadIdx.x);
}

int example2()
{
	warpTest << <5, 32 >> > ();
	// ������� ������� ����� �������
	getchar();
	return 0;
}