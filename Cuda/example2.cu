#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
__global__ void HelloWorld()
{
	printf("Hello world, %d, %d\n", blockIdx.x,
		threadIdx.x);
}

int example2()
{
	HelloWorld << <4, 4 >> > ();
	// ������� ������� ����� �������
	getchar();
	return 0;
}