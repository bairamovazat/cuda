#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
using namespace std;

// ����
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int classWork1() {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	//���������� ����������������� �� ����������
	printf("���������� ����������������� �� ����������: %d\n", deviceProp.multiProcessorCount);
	//���������� ������ �������� �� ���������� � ������
	printf("���������� ������ �������� �� ���������� � ������ : %d MB\n", deviceProp.totalGlobalMem / 1024 / 1024);
	//�������� ������� � ����������
	printf("�������� ������� � ����������: %d\n", deviceProp.clockRate);
	//������� �������� ������� ������ � ����������
	printf("������� �������� ������� ������ � ����������: %d\n", deviceProp.memoryClockRate);
	//������ ���� ���������� ������ � �����
	printf("������ ���� ���������� ������ � �����: %d\n", deviceProp.memoryBusWidth);

	// ���������� �� CPU
	int a, b, c;

	// ���������� �� GPU
	int *dev_a, *dev_b, *dev_c;
	int size = sizeof(int); //�����������
	// �������� ������ �� GPU
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	// ������������� ����������
	a = 2;
	b = 7;
	// ����������� ���������� � CPU �� GPU
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
	// ����� ����
	add << < 1, 1 >> > (dev_a, dev_b, dev_c);
	// ����������� ���������� ������ ���� � GPU �� CPU
	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
	// ����� ����������
	printf("%d + %d = %d\n", a, b, c);
	// �������� ������ �� GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}